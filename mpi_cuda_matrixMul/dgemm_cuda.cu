/*
 *
 */

/* Matrix multiplication: C = A * B.
 * Host code.
 * performant generic kernel for matrix multiplication.
 *
 */

#include <hipblas.h>

#include <hip/hip_runtime.h>
#include "dgemm_cuda.h"
#include <stdio.h>
#include <stdlib.h>
#include <string>
#include <assert.h>
#include <math.h>

#include <fstream>
#include <vector>
#include <iostream>
#include <algorithm>

typedef struct {
   double *local_A;
   double *local_B;
   double *local_C;
   int m;
   int n_threads;
   int tid;
   int rank;
} ThreadsInfo;


////////////////////////////////////////////////////////////////////////////////
// These are CUDA Helper functions

    // This will output the proper CUDA error strings in the event that a CUDA host call returns an error
    #define checkCudaErrors(err)           __checkCudaErrors (err, __FILE__, __LINE__)

    inline void __checkCudaErrors( hipError_t err, const char *file, const int line )
    {
        if( hipSuccess != err) {
		    fprintf(stderr, "%s(%i) : CUDA Runtime API error %d: %s.\n",
                    file, line, (int)err, hipGetErrorString( err ) );
            exit(-1);
        }
    }

    // This will output the proper error string when calling hipGetLastError
    #define getLastCudaError(msg)      __getLastCudaError (msg, __FILE__, __LINE__)

    inline void __getLastCudaError( const char *errorMessage, const char *file, const int line )
    {
        hipError_t err = hipGetLastError();
        if( hipSuccess != err) {
            fprintf(stderr, "%s(%i) : getLastCudaError() CUDA error : %s : (%d) %s.\n",
                    file, line, errorMessage, (int)err, hipGetErrorString( err ) );
            exit(-1);
        }
    }

    // General GPU Device CUDA Initialization
    int gpuDeviceInit(int devID)
    {
        int deviceCount;
        checkCudaErrors(hipGetDeviceCount(&deviceCount));
        if (deviceCount == 0) {
            fprintf(stderr, "gpuDeviceInit() CUDA error: no devices supporting CUDA.\n");
            exit(-1);
        }
        if (devID < 0) 
            devID = 0;
        if (devID > deviceCount-1) {
            fprintf(stderr, "\n");
            fprintf(stderr, ">> %d CUDA capable GPU device(s) detected. <<\n", deviceCount);
            fprintf(stderr, ">> gpuDeviceInit (-device=%d) is not a valid GPU device. <<\n", devID);
            fprintf(stderr, "\n");
            return -devID;
        }

        hipDeviceProp_t deviceProp;
        checkCudaErrors( hipGetDeviceProperties(&deviceProp, devID) );
        if (deviceProp.major < 1) {
            fprintf(stderr, "gpuDeviceInit(): GPU device does not support CUDA.\n");
            exit(-1);                                                  \
        }

        checkCudaErrors( hipSetDevice(devID) );
        //printf("> gpuDeviceInit() CUDA device [%d]: %s\n", devID, deviceProp.name);
        return devID;
    }

    // This function returns the best GPU (with maximum GFLOPS)
    // Initialization code to find the best CUDA Device
// end of CUDA Helper Functions

////////////////////////////////////////////////////////////////////////////////
// declaration, forward
void randomInit(float*, int);

void inline checkError(hipblasStatus_t status, const char* msg)
{
    if(status != HIPBLAS_STATUS_SUCCESS){
        printf(msg);
        exit(-1);
    }
}

////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////
////////////////////////////////////////////////////////////////////////////////
//! Run a simple test for CUDA
////////////////////////////////////////////////////////////////////////////////

extern "C" void doCUDA(void *ptr)
{

    ThreadsInfo *threads_info = (ThreadsInfo *)ptr;                                                           
    int size = threads_info->m;                   
    int devID;
    hipDeviceProp_t props;

    // get number of SMs on this GPU
    checkCudaErrors(hipGetDevice(&devID));
    checkCudaErrors(hipGetDeviceProperties(&props, devID));

    // use a larger block size for Fermi and above
    int block_size = (props.major < 2) ? 16 : 32;
    srand(2006);
    // Optional Command-line multiplier for matrix sizes
    unsigned int uiWA, uiHA, uiWB, uiHB, uiWC, uiHC;
	
	uiWA = size;
	uiHA = size;
	uiWB = size;
	uiHB = size;
	uiWC = size;
	uiHC = size;

    // allocate host memory for matrices A and B
    unsigned int size_A = uiWA * uiHA;
    unsigned int mem_size_A = sizeof(float) * size_A;
    float* h_A = (float*)malloc(mem_size_A);
    unsigned int size_B = uiWB * uiHB;
    unsigned int mem_size_B = sizeof(float) * size_B;
    float* h_B = (float*)malloc(mem_size_B);

    // initialize host memory
    randomInit(h_A, size_A);
    randomInit(h_B, size_B);
    
    // allocate device memory
    float* d_A, *d_B, *d_C;
    unsigned int size_C = uiWC * uiHC;
    unsigned int mem_size_C = sizeof(float) * size_C;

    // allocate host memory for the result
    float* h_C      = (float*) malloc(mem_size_C);
    float* h_CUBLAS = (float*) malloc(mem_size_C);

    checkCudaErrors(hipMalloc((void**) &d_A, mem_size_A));
    checkCudaErrors(hipMalloc((void**) &d_B, mem_size_B));

    // copy host memory to device
    checkCudaErrors(hipMemcpy(d_A, h_A, mem_size_A, hipMemcpyHostToDevice) );
    checkCudaErrors(hipMemcpy(d_B, h_B, mem_size_B, hipMemcpyHostToDevice) );
    checkCudaErrors(hipMalloc((void**) &d_C, mem_size_C));
   
    //setup execution parameters
    dim3 threads(block_size, block_size);
    dim3 grid(uiWC / threads.x, uiHC / threads.y);
    
	// CUBLAS version 2.0
        hipblasHandle_t handle;
        checkError(hipblasCreate(&handle), "hipblasCreate() error!\n");
        const float alpha = 1.0f;
        const float beta = 0.0f;
	hipblasStatus_t ret = hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, uiWB, uiHA, uiWA, &alpha, d_B, uiWB, d_A, uiWA, &beta, d_C, uiWA);
        checkError(ret, "cublas Sgemm returned an error!\n");

	getLastCudaError("CUBLAS Kernel execution failed");
	hipDeviceSynchronize();

	checkCudaErrors(hipMemcpy(h_CUBLAS, d_C, mem_size_C, hipMemcpyDeviceToHost) );
        checkError(hipblasDestroy(handle), "hipblasDestroy() error!\n");
        hipDeviceSynchronize();
	
	getLastCudaError("CUDA matrixMul Kernel execution failed");
	checkCudaErrors(hipMemcpy(h_C, d_C, mem_size_C, hipMemcpyDeviceToHost) );
    free(h_A);
    free(h_B);
    free(h_C);
    checkCudaErrors(hipFree(d_A));
    checkCudaErrors(hipFree(d_B));
    checkCudaErrors(hipFree(d_C));
    hipDeviceReset();
}

// Allocates a matrix with random float entries.
void randomInit(float* data, int size)
{
    for (int i = 0; i < size; ++i)
        data[i] = rand() / (float)RAND_MAX;
}

