#include "hip/hip_runtime.h"
#include <mpi.h>
#include <omp.h>
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <cutil.h>
#include <cmeansMPI.h>
#include <math.h>
#include <time.h>
#include <string.h>
#include <float.h>
#include <cmeansMPI_kernel.cu>
#include "MDL.h"

void printCudaError() {
    hipError_t error = hipGetLastError();
    if(error != hipSuccess) {
        printf("%s\n",hipGetErrorString(error));
    }
}

typedef struct {
    hipEvent_t start;
    hipEvent_t stop;
    float* et;
} cudaTimer_t;

void createTimer(cudaTimer_t* timer) {
    #pragma omp critical (create_timer) 
    {
        hipEventCreate(&(timer->start));
        hipEventCreate(&(timer->stop));
        timer->et = (float*) malloc(sizeof(float));
        *(timer->et) = 0.0f;
    }
}

void deleteTimer(cudaTimer_t timer) {
    #pragma omp critical (delete_timer) 
    {
        hipEventDestroy(timer.start);
        hipEventDestroy(timer.stop);
        free(timer.et);
    }
}

void startTimer(cudaTimer_t timer) {
    hipEventRecord(timer.start,0);
}

void stopTimer(cudaTimer_t timer) {
    hipEventRecord(timer.stop,0);
    hipEventSynchronize(timer.stop);
    float tmp;
    hipEventElapsedTime(&tmp,timer.start,timer.stop);
    *(timer.et) += tmp;
}

float getTimerValue(cudaTimer_t timer) {
    return *(timer.et);
}

/************************************************************************/
/* C-means Main                                                            */
/************************************************************************/
int main(int argc, char* argv[])
{
    int rank, num_nodes, len, provided;
    char name[MPI_MAX_PROCESSOR_NAME];

    MPI_Init_thread(&argc,&argv,MPI_THREAD_MULTIPLE,&provided);
    MPI_Comm_size(MPI_COMM_WORLD,&num_nodes);
    MPI_Comm_rank(MPI_COMM_WORLD,&rank);
    MPI_Get_processor_name(name, &len);
    printf("Hello world from node %d of %d on %s\n",rank,num_nodes,name);

    unsigned int timer_io; // Timer for I/O, such as reading FCS file and outputting result files
    unsigned int timer_total; // Total time
    unsigned int timer_main_cpu; // Total time
   
    cutCreateTimer(&timer_io);
    cutCreateTimer(&timer_total);
    cutCreateTimer(&timer_main_cpu);
    
    // determine the number of CUDA capable GPUs
    int num_gpus = 0;       // number of CUDA GPUs
    hipGetDeviceCount(&num_gpus);
    if(num_gpus < 1)
    {
        printf("no CUDA capable devices were detected\n");
        return 1;
    }

    // display CPU and GPU configuration
    printf("number of host CPUs:\t%d\n", omp_get_num_procs());
    printf("number of CUDA devices:\t%d\n", num_gpus);
    for(int i = 0; i < num_gpus; i++)
    {
        hipDeviceProp_t dprop;
        hipGetDeviceProperties(&dprop, i);
        printf("   %d: %s\n", i, dprop.name);
    }
    printf("---------------------------\n");
    
    int total_num_gpus = num_gpus * num_nodes;
    
    cutStartTimer(timer_total);
    
    // [program name]  [data file]
    if(argc != 2){
        printf("Usage Error: must supply data file. e.g. programe_name @opt(flags) file.in\n");
        return 1;
    }

    cutStartTimer(timer_io);
    float* myEvents;

    int elements_per_node, elements_being_sent;
    elements_per_node = NUM_EVENTS / total_num_gpus * num_gpus * NUM_DIMENSIONS;
    
    // Root reads input from file and distributes to each node
    if(rank == 0) {
        myEvents = ParseSampleInput(argv[1]);
        MPI_Request* requests = (MPI_Request*) malloc(sizeof(MPI_Request)*num_nodes);
        MPI_Status s;
        // Send everything asynchronously
        for(int i=1; i < num_nodes; i++) {
            elements_being_sent = elements_per_node;
            if(i == num_nodes-1) { // boundary condition
                elements_being_sent += (NUM_EVENTS % total_num_gpus)*NUM_DIMENSIONS;
            }
            MPI_Isend(&(myEvents[elements_per_node*i]),elements_being_sent,MPI_FLOAT,i,1,MPI_COMM_WORLD,&requests[i]);
            //MPI_Send(&(myEvents[elements_per_node*i]),elements_being_sent,MPI_FLOAT,i,1,MPI_COMM_WORLD);
        }
        // Wait for the Isends to complete
        for(int i=1; i < num_nodes; i++) {
            MPI_Wait(&requests[i],&s);
        }
        free(requests);
        elements_being_sent = elements_per_node; // so that its set properly for the root 
    } else {
        myEvents = (float*) malloc(sizeof(float)*NUM_DIMENSIONS*NUM_EVENTS);
        elements_being_sent = elements_per_node;
        if(rank == num_nodes-1) { // boundary condition
            elements_being_sent += (NUM_EVENTS % total_num_gpus)*NUM_DIMENSIONS;
        }
        MPI_Status s;
        MPI_Recv(&(myEvents[elements_per_node*rank]),elements_being_sent,MPI_FLOAT,0,1,MPI_COMM_WORLD,&s);
    }
    MPI_Barrier(MPI_COMM_WORLD); 
    cutStopTimer(timer_io);
    
    cutStartTimer(timer_main_cpu);
    //srand((unsigned)(time(0)));
    srand(42);
    
    // Allocate arrays for the cluster centers
    float* myClusters = (float*)malloc(sizeof(float)*NUM_CLUSTERS*NUM_DIMENSIONS);
    float* newClusters = (float*)malloc(sizeof(float)*NUM_CLUSTERS*NUM_DIMENSIONS);

    // Select random cluster centers
    // double t1,t2;
    generateInitialClusters(myClusters, myEvents);


    // Create an array of arrays for temporary cluster centers from each GPU
    float** tempClusters = (float**) malloc(sizeof(float*)*num_gpus);
    float** tempDenominators = (float**) malloc(sizeof(float*)*num_gpus);
    for(int i=0; i < num_gpus; i++) {
        tempClusters[i] = (float*) malloc(sizeof(float)*NUM_CLUSTERS*NUM_DIMENSIONS);
        tempDenominators[i] = (float*) malloc(sizeof(float)*NUM_CLUSTERS);
        memcpy(tempClusters[i],myClusters,sizeof(float)*NUM_CLUSTERS*NUM_DIMENSIONS);
    }
    // Create an array of arrays for temporary Q matrix pieces from each GPU
    float** q_matrices = (float**) malloc(sizeof(float*)*num_gpus);
    // Create an array for the final Q matrix
    float* q_matrix = (float*) malloc(sizeof(float)*NUM_CLUSTERS*NUM_CLUSTERS);
    
    float diff; // used to track difference in cluster centers between iterations

    // Transpose the events matrix
    float* transposedEvents = (float*)malloc(sizeof(float)*NUM_EVENTS*NUM_DIMENSIONS);
    for(int i=0; i<NUM_EVENTS; i++) {
        for(int j=0; j<NUM_DIMENSIONS; j++) {
            transposedEvents[j*NUM_EVENTS+i] = myEvents[i*NUM_DIMENSIONS+j];
        }
    }

    float* memberships = (float*) malloc(sizeof(float)*NUM_CLUSTERS*NUM_EVENTS);
    int* finalClusterConfig;
    cutStopTimer(timer_main_cpu);
   
    ////////////////////////////////////////////////////////////////
    // run as many CPU threads as there are CUDA devices
    //num_gpus = 1;
    //omp_set_num_threads(num_gpus);  // create as many CPU threads as there are CUDA devices
    #pragma omp parallel shared(myClusters,diff,tempClusters,tempDenominators,memberships,finalClusterConfig)
    {
        cudaTimer_t timer_memcpy; // Timer for GPU <---> CPU memory copying
        cudaTimer_t timer_cpu; // Timer for processing on CPU
        cudaTimer_t timer_gpu; // Timer for kernels on the GPU
        cudaTimer_t timer_mpi; // Timer for MPI
        
        unsigned int tid = omp_get_thread_num();
        unsigned int num_cpu_threads = omp_get_num_threads();
        int gpu_num = rank*num_gpus+tid;
        printf("hello from thread %d of %d\n",tid,num_cpu_threads);

        // set and check the CUDA device for this CPU thread
        int gpu_id = -1;
        hipSetDevice(tid % num_gpus);        // "% num_gpus" allows more CPU threads than GPU devices
        hipGetDevice(&gpu_id);
       
        #pragma omp barrier
 
        createTimer(&timer_memcpy);
        createTimer(&timer_cpu);
        createTimer(&timer_gpu);
        createTimer(&timer_mpi);

        printf("CPU thread %d (of %d) uses CUDA device %d\n", tid, num_cpu_threads, gpu_id);
        
        // Compute starting/finishing indexes for the events for each gpu
        int events_per_gpu = NUM_EVENTS / total_num_gpus;
        int my_num_events = events_per_gpu;
        if(gpu_num == (total_num_gpus-1)) {
            my_num_events += NUM_EVENTS % total_num_gpus;
        }

        startTimer(timer_memcpy);
        float* d_distanceMatrix;
        CUDA_SAFE_CALL(hipMalloc((void**)&d_distanceMatrix, sizeof(float)*my_num_events*NUM_CLUSTERS));
        #if !LINEAR
            float* d_memberships;
            CUDA_SAFE_CALL(hipMalloc((void**)&d_memberships, sizeof(float)*my_num_events*NUM_CLUSTERS));
        #endif
        float* d_E;
        CUDA_SAFE_CALL(hipMalloc((void**)&d_E, sizeof(float)*my_num_events*NUM_DIMENSIONS));
        float* d_C;
        CUDA_SAFE_CALL(hipMalloc((void**)&d_C, sizeof(float)*NUM_CLUSTERS*NUM_DIMENSIONS));
        float* d_nC;
        CUDA_SAFE_CALL(hipMalloc((void**)&d_nC, sizeof(float)*NUM_CLUSTERS*NUM_DIMENSIONS));
        float* d_denoms;
        CUDA_SAFE_CALL(hipMalloc((void**)&d_denoms, sizeof(float)*NUM_CLUSTERS));

        int size = sizeof(float)*NUM_DIMENSIONS*my_num_events;

        // Copying the transposed data is trickier since it's not all contigious for the relavant events
        float* temp_fcs_data = (float*) malloc(size);
        for(int d=0; d < NUM_DIMENSIONS; d++) {
            memcpy(&temp_fcs_data[d*my_num_events],&transposedEvents[d*NUM_EVENTS + gpu_num*events_per_gpu],sizeof(float)*my_num_events);
        }
        CUDA_SAFE_CALL(hipMemcpy( d_E, temp_fcs_data, size,hipMemcpyHostToDevice) );
        hipDeviceSynchronize();
        free(temp_fcs_data);

        size = sizeof(float)*NUM_DIMENSIONS*NUM_CLUSTERS;
        CUDA_SAFE_CALL(hipMemcpy(d_C, myClusters, size, hipMemcpyHostToDevice));
        stopTimer(timer_memcpy);
        
        printf("Starting C-means\n");
        int iterations = 0;
        

        int num_blocks_distance = my_num_events / NUM_THREADS_DISTANCE;
        if(my_num_events % NUM_THREADS_DISTANCE) {
            num_blocks_distance++;
        }
        int num_blocks_membership = my_num_events / NUM_THREADS_MEMBERSHIP;
        if(my_num_events % NUM_THREADS_DISTANCE) {
            num_blocks_membership++;
        }
        int num_blocks_update = NUM_CLUSTERS / NUM_CLUSTERS_PER_BLOCK;
        if(NUM_CLUSTERS % NUM_CLUSTERS_PER_BLOCK) {
            num_blocks_update++;
        }

        do{
            cudaTimer_t timer;
            createTimer(&timer);
            startTimer(timer);

            size = sizeof(float)*NUM_DIMENSIONS*NUM_CLUSTERS;

            // Copy the cluster centers to the GPU
            startTimer(timer_memcpy);
            CUDA_SAFE_CALL(hipMemcpy(d_C, myClusters, size, hipMemcpyHostToDevice));
            stopTimer(timer_memcpy);
            

            startTimer(timer_gpu);
            DEBUG("Launching ComputeDistanceMatrix kernel\n");
            ComputeDistanceMatrix<<< dim3(num_blocks_distance,NUM_CLUSTERS), NUM_THREADS_DISTANCE  >>>(d_C, d_E, d_distanceMatrix, my_num_events);
            #if LINEAR
                // O(M) membership kernel
                DEBUG("Launching ComputeMembershipMatrixLinear kernel\n");
                ComputeMembershipMatrixLinear<<< num_blocks_membership, NUM_THREADS_MEMBERSHIP  >>>(d_distanceMatrix, my_num_events);
                DEBUG("Launching UpdateClusterCentersGPU kernel\n");
                //UpdateClusterCentersGPU<<< dim3(NUM_CLUSTERS,NUM_DIMENSIONS), NUM_THREADS_UPDATE >>>(d_C, d_E, d_nC, d_distanceMatrix, d_denoms, my_num_events);
                //UpdateClusterCentersGPU2<<< dim3(num_blocks_update,NUM_DIMENSIONS), NUM_THREADS_UPDATE >>>(d_C, d_E, d_nC, d_distanceMatrix, my_num_events);
                UpdateClusterCentersGPU3<<< dim3(NUM_DIMENSIONS,num_blocks_update), NUM_THREADS_UPDATE >>>(d_C, d_E, d_nC, d_distanceMatrix, my_num_events);
                ComputeClusterSizes<<< NUM_CLUSTERS, 512 >>>( d_distanceMatrix, d_denoms, my_num_events);
            #else
                // O(M^2) membership kernel
                DEBUG("Launching ComputeMembershipMatrix kernel\n");
                ComputeMembershipMatrix<<< dim3(num_blocks_membership,NUM_CLUSTERS), NUM_THREADS_MEMBERSHIP  >>>(d_distanceMatrix, d_memberships, my_num_events);
                DEBUG("Launching UpdateClusterCentersGPU kernel\n");
                //UpdateClusterCentersGPU<<< dim3(NUM_CLUSTERS,NUM_DIMENSIONS), NUM_THREADS_UPDATE >>>(d_C, d_E, d_nC, d_memberships, d_denoms, my_num_events);
                //UpdateClusterCentersGPU2<<< dim3(num_blocks_update,NUM_DIMENSIONS), NUM_THREADS_UPDATE >>>(d_C, d_E, d_nC, d_memberships, my_num_events);
                UpdateClusterCentersGPU3<<< dim3(NUM_DIMENSIONS,num_blocks_update), NUM_THREADS_UPDATE >>>(d_C, d_E, d_nC, d_memberships, my_num_events);
                ComputeClusterSizes<<< NUM_CLUSTERS, 512 >>>( d_memberships, d_denoms, my_num_events );
            #endif

            hipDeviceSynchronize();
            printCudaError();
            
            stopTimer(timer_gpu);
            
            // Copy partial centers and denominators to host
            startTimer(timer_memcpy);
            hipMemcpy(tempClusters[tid], d_nC, sizeof(float)*NUM_CLUSTERS*NUM_DIMENSIONS, hipMemcpyDeviceToHost);
            hipMemcpy(tempDenominators[tid], d_denoms, sizeof(float)*NUM_CLUSTERS, hipMemcpyDeviceToHost);
            printCudaError();
            stopTimer(timer_memcpy);
            
            stopTimer(timer);
            float thisTime = getTimerValue(timer);
            DEBUG("Processing time for GPU %d: %f (ms) \n", tid, thisTime);
            deleteTimer(timer);

        
            #pragma omp barrier
            #pragma omp master
            {
                startTimer(timer_cpu);
                // Sum up the partial cluster centers (numerators)
                for(int i=1; i < num_gpus; i++) {
                    for(int c=0; c < NUM_CLUSTERS; c++) {
                        for(int d=0; d < NUM_DIMENSIONS; d++) {
                            tempClusters[0][c*NUM_DIMENSIONS+d] += tempClusters[i][c*NUM_DIMENSIONS+d];
                        }
                    }
                }

                // Sum up the denominator for each cluster
                for(int i=1; i < num_gpus; i++) {
                    for(int c=0; c < NUM_CLUSTERS; c++) {
                        tempDenominators[0][c] += tempDenominators[i][c];
                    }
                }
                stopTimer(timer_cpu);

                DEBUG("Reducing cluster values\n");
                startTimer(timer_mpi);
                if(rank == 0) {
                    MPI_Reduce(MPI_IN_PLACE,tempClusters[0],NUM_DIMENSIONS*NUM_CLUSTERS,MPI_FLOAT,MPI_SUM,0,MPI_COMM_WORLD);
                    //MPI_Reduce(tempClusters[0],tempClusters[1],NUM_DIMENSIONS*NUM_CLUSTERS,MPI_FLOAT,MPI_SUM,0,MPI_COMM_WORLD);
                    //memcpy(tempClusters[0],tempClusters[1],sizeof(float)*NUM_DIMENSIONS*NUM_CLUSTERS);
                    MPI_Reduce(MPI_IN_PLACE,tempDenominators[0],NUM_CLUSTERS,MPI_FLOAT,MPI_SUM,0,MPI_COMM_WORLD);
                    //MPI_Reduce(tempDenominators[0],tempDenominators[1],NUM_CLUSTERS,MPI_FLOAT,MPI_SUM,0,MPI_COMM_WORLD);
                    //memcpy(tempDenominators[0],tempDenominators[1],sizeof(float)*NUM_CLUSTERS);
                } else {
                    MPI_Reduce(tempClusters[0],0,NUM_DIMENSIONS*NUM_CLUSTERS,MPI_FLOAT,MPI_SUM,0,MPI_COMM_WORLD);
                    MPI_Reduce(tempDenominators[0],0,NUM_CLUSTERS,MPI_FLOAT,MPI_SUM,0,MPI_COMM_WORLD);
                }
                MPI_Barrier(MPI_COMM_WORLD); // not sure if neccesary...
                stopTimer(timer_mpi);

                startTimer(timer_cpu);
                // Divide to get the final clusters
                if(rank == 0) {
                    for(int c=0; c < NUM_CLUSTERS; c++) {
                        for(int d=0; d < NUM_DIMENSIONS; d++) {
                            tempClusters[0][c*NUM_DIMENSIONS+d] /= tempDenominators[0][c];
                        }
                    }
                }
                stopTimer(timer_cpu);
                startTimer(timer_mpi);
                DEBUG("Broadcasting Cluster Values\n");
                MPI_Bcast(tempClusters[0],NUM_DIMENSIONS*NUM_CLUSTERS,MPI_FLOAT,0,MPI_COMM_WORLD);
                MPI_Barrier(MPI_COMM_WORLD);
                stopTimer(timer_mpi);

                startTimer(timer_cpu);
                diff = 0.0;
                for(int i=0; i < NUM_CLUSTERS; i++){
                    DEBUG("GPU %d, Cluster %d: ",tid,i);
                    for(int k = 0; k < NUM_DIMENSIONS; k++){
                        DEBUG("%f ",tempClusters[tid][i*NUM_DIMENSIONS + k]);
                        diff += fabs(myClusters[i*NUM_DIMENSIONS + k] - tempClusters[tid][i*NUM_DIMENSIONS + k]);
                    }
                    DEBUG("\n");
                }
                memcpy(myClusters,tempClusters[tid],sizeof(float)*NUM_DIMENSIONS*NUM_CLUSTERS);
                DEBUG("Diff = %f\n", diff);
                DEBUG("Done with iteration #%d\n", iterations);
                stopTimer(timer_cpu);
            }
            #pragma omp barrier
            iterations++;
            DEBUG("\n");
        } while(iterations < MIN_ITERS || (abs(diff) > THRESHOLD && iterations < MAX_ITERS)); 

        #pragma omp master
        {
            if(rank == 0) {
                printf("Iterations: %d\n",iterations);
            }
        }
        #if ENABLE_OUTPUT
            // Compute final membership vaues
            startTimer(timer_gpu);
            #if LINEAR
                // O(M)
                ComputeDistanceMatrix<<< dim3(num_blocks_distance,NUM_CLUSTERS), NUM_THREADS_DISTANCE  >>>(d_C, d_E, d_distanceMatrix, my_num_events);
                ComputeNormalizedMembershipMatrixLinear<<< num_blocks_membership, NUM_THREADS_MEMBERSHIP >>>(d_distanceMatrix,my_num_events);
            #else
                // O(M^2)
                ComputeNormalizedMembershipMatrix<<< dim3(num_blocks_membership,NUM_CLUSTERS), NUM_THREADS_MEMBERSHIP  >>>(d_distanceMatrix, d_memberships, my_num_events);
            #endif
            stopTimer(timer_gpu);

            // Copy memberships from the GPU
            float* temp_memberships = (float*) malloc(sizeof(float)*my_num_events*NUM_CLUSTERS);
            startTimer(timer_memcpy);
            #if LINEAR
                hipMemcpy(temp_memberships,d_distanceMatrix,sizeof(float)*my_num_events*NUM_CLUSTERS,hipMemcpyDeviceToHost);
            #else
                hipMemcpy(temp_memberships,d_memberships,sizeof(float)*my_num_events*NUM_CLUSTERS,hipMemcpyDeviceToHost);
            #endif
            stopTimer(timer_memcpy);

            startTimer(timer_cpu);
            for(int c=0; c < NUM_CLUSTERS; c++) {
                memcpy(&(memberships[c*NUM_EVENTS+gpu_num*events_per_gpu]),&(temp_memberships[c*my_num_events]),sizeof(float)*my_num_events);
            }
            stopTimer(timer_cpu);
            #pragma omp barrier
            #pragma omp master
            {
                startTimer(timer_cpu);
                // First transpose the memberships, makes it easier to gather the results between nodes
                float* temp = (float*) malloc(sizeof(float)*NUM_EVENTS*NUM_CLUSTERS);
                for(int e=0; e < NUM_EVENTS; e++) {
                    for(int c=0; c < NUM_CLUSTERS; c++) {
                        temp[e*NUM_CLUSTERS+c] = memberships[c*NUM_EVENTS+e];
                    }
                }
                memcpy(memberships,temp,sizeof(float)*NUM_EVENTS*NUM_CLUSTERS);
                stopTimer(timer_cpu);
                // Gather memberships on root        
                startTimer(timer_mpi);
                int memberships_being_sent, memberships_per_node;
                memberships_per_node = events_per_gpu*num_gpus*NUM_CLUSTERS;
                if(rank == 0) {
                    for(int i=1; i < num_nodes; i++) {
                        memberships_being_sent = memberships_per_node;
                        if(i == num_nodes-1) { // boundary condition
                            memberships_being_sent += (NUM_EVENTS % total_num_gpus)*NUM_CLUSTERS;
                        }
                        MPI_Status s;
                        MPI_Recv(&(temp[memberships_per_node*i]),memberships_being_sent,MPI_FLOAT,i,1,MPI_COMM_WORLD,&s);
                    }
                } else {
                    memberships_being_sent = memberships_per_node;
                    if(rank == num_nodes-1) { // boundary condition
                        memberships_being_sent += (NUM_EVENTS % total_num_gpus)*NUM_CLUSTERS;
                    }
                    MPI_Send(&(memberships[memberships_per_node*rank]),memberships_being_sent,MPI_FLOAT,0,1,MPI_COMM_WORLD);
                }
                MPI_Barrier(MPI_COMM_WORLD);
                stopTimer(timer_mpi);
                // Tranpose the memberships again to get original ordering
                startTimer(timer_cpu);
                if(rank == 0) {
                    for(int e=0; e < NUM_EVENTS; e++) {
                        for(int c=0; c<NUM_CLUSTERS; c++) {
                            memberships[c*NUM_EVENTS+e] = temp[e*NUM_CLUSTERS+c];
                        }
                    }    
                }
                free(temp);
                stopTimer(timer_cpu);
            }
            #pragma omp barrier
            free(temp_memberships);
        #endif // #if ENABLE_OUTPUT

        if(tid == 0) {        
            if(abs(diff) > THRESHOLD){
                PRINT("Warning: c-means did not converge to the %f threshold provided\n", THRESHOLD);
            }
            PRINT("C-means complete\n");
        }
        
        #pragma omp barrier // sync threads 
       
        #if !ENABLE_MDL
            if(tid == 0) {
                // Don't attempt MDL, save all clusters 
                finalClusterConfig = (int*) malloc(sizeof(int)*NUM_CLUSTERS);
                memset(finalClusterConfig,1,sizeof(int)*NUM_CLUSTERS);
            }
        #else
            PRINT("Calculating Q Matrix Section %d\n",tid);
           
            // Copy the latest clusters to the device 
            //  (the current ones on the device are 1 iteration old) 
            startTimer(timer_memcpy);
            CUDA_SAFE_CALL(hipMemcpy(d_C, myClusters, size, hipMemcpyHostToDevice));
            stopTimer(timer_memcpy);
            
            // Build Q matrix, each gpu handles NUM_DIMENSIONS/num_gpus rows of the matrix
            q_matrices[tid] = BuildQGPU(d_E, d_C, d_distanceMatrix, &mdlTime, tid, num_gpus, my_num_events);
            
            #pragma omp barrier // sync threads
            
            if(tid == 0) {
                // Combine the partial matrices
                int num_matrix_elements = NUM_CLUSTERS*(NUM_CLUSTERS/num_gpus);
                for(int i=0; i < num_gpus; i++) {
                    float* q_matrix_ptr = (float*) q_matrix+i*num_matrix_elements;
                    float* q_matrices_ptr = (float*) q_matrices[i]+i*num_matrix_elements;
                    memcpy(q_matrix_ptr,q_matrices_ptr,sizeof(float)*num_matrix_elements);   
                    free(q_matrices[i]);
                }
                startTimer(timer_cpu);
                DEBUG("Searching for optimal configuration...\n");
                finalClusterConfig = TabuSearch(q_matrix, argv[1]);
                stopTimer(timer_cpu);

                DEBUG("Q Matrix:\n");
                for(int row=0; row < NUM_CLUSTERS; row++) {
                    for(int col=0; col < NUM_CLUSTERS; col++) {
                        DEBUG("%.2e ",q_matrix[row*NUM_CLUSTERS+col]);
                    }
                    DEBUG("\n");
                }
                
                free(q_matrix);
            }
            mdlTime /= 1000.0; // CUDA timer returns time in milliseconds, normalize to seconds
        #endif

        fflush(stdout);
        #pragma omp barrier

        #pragma omp master
        { 
            printf("\n\n"); 
            printf("Node %d: Thread %d: GPU memcpy Time (ms): %f\n",rank,tid,getTimerValue(timer_memcpy));
            printf("Node %d: Thread %d: CPU processing Time (ms): %f\n",rank,tid,getTimerValue(timer_cpu));
            printf("Node %d: Thread %d: GPU processing Time (ms): %f\n",rank,tid,getTimerValue(timer_gpu));
            printf("Node %d: Thread %d: MPI Time (ms): %f\n",rank,tid,getTimerValue(timer_mpi));
        }        

        #if !CPU_ONLY
            CUDA_SAFE_CALL(hipFree(d_E));
            CUDA_SAFE_CALL(hipFree(d_C));
            CUDA_SAFE_CALL(hipFree(d_nC));
        #endif
    
        #pragma omp barrier
        DEBUG("Thread %d done.\n",tid);
    } // end of omp_parallel block
    
    cutStartTimer(timer_io);

    if(rank == 0) {
        PRINT("Final Clusters are:\n");
        int newCount = 0;
        for(int i = 0; i < NUM_CLUSTERS; i++){
            if(finalClusterConfig[i]){
                for(int j = 0; j < NUM_DIMENSIONS; j++){
                    newClusters[newCount * NUM_DIMENSIONS + j] = myClusters[i*NUM_DIMENSIONS + j];
                    PRINT("%.3f\t", myClusters[i*NUM_DIMENSIONS + j]);
                }
                newCount++;
                PRINT("\n");
            }
        }
        
        #if ENABLE_OUTPUT 
            ReportSummary(newClusters, newCount, argv[1]);
            ReportResults(myEvents, memberships, newCount, argv[1]);
        #endif
    }

    cutStopTimer(timer_io);
    cutStopTimer(timer_total);
   
    if(rank == 0) { 
        printf("Total Time (ms): %f\n",cutGetTimerValue(timer_total));
        printf("I/O Time (ms): %f\n",cutGetTimerValue(timer_io));
        printf("Main Thread CPU Time (ms): %f\n",cutGetTimerValue(timer_main_cpu));
        printf("\n\n");
    } 
    
    free(newClusters);
    free(myClusters);
    free(myEvents);
    free(transposedEvents);
    MPI_Finalize();
    return 0;
}


void generateInitialClusters(float* clusters, float* events){
    int seed;
    for(int i = 0; i < NUM_CLUSTERS; i++){
        //seed = i * NUM_EVENTS / NUM_CLUSTERS;
        seed = rand() % NUM_EVENTS;
        for(int j = 0; j < NUM_DIMENSIONS; j++){
            clusters[i*NUM_DIMENSIONS + j] = events[seed*NUM_DIMENSIONS + j];
        }
    }
    
}

float* readBIN(char* f) {
    FILE* fin = fopen(f,"rb");
    int nevents,ndims;
    fread(&nevents,4,1,fin);
    fread(&ndims,4,1,fin);
    int num_elements = (ndims)*(nevents);
    printf("Number of rows: %d\n",nevents);
    printf("Number of cols: %d\n",ndims);
    float* data = (float*) malloc(sizeof(float)*num_elements);
    fread(data,sizeof(float),num_elements,fin);
    fclose(fin);
    return data;
}


float* readCSV(char* filename) {
    FILE* myfile = fopen(filename, "r");
    if(myfile == NULL){
        printf("Error: File DNE\n");
        return NULL;
    }
    char myline[1024];
    
    float* retVal = (float*)malloc(sizeof(float)*NUM_EVENTS*NUM_DIMENSIONS);
    myfile = fopen(filename, "r");
    #if LINE_LABELS
        fgets(myline, 1024, myfile);
        for(int i = 0; i < NUM_EVENTS; i++){
            fgets(myline, 1024, myfile);
            retVal[i*NUM_DIMENSIONS] = (float)atof(strtok(myline, DELIMITER));
            for(int j = 1; j < NUM_DIMENSIONS; j++){
                retVal[i*NUM_DIMENSIONS + j] = (float)atof(strtok(NULL, DELIMITER));
            }
        }
    #else
        for(int i = 0; i < NUM_EVENTS; i++){
            fgets(myline, 1024, myfile);
            retVal[i*NUM_DIMENSIONS] = (float)atof(strtok(myline, DELIMITER));
            for(int j = 1; j < NUM_DIMENSIONS; j++){
                retVal[i*NUM_DIMENSIONS + j] = (float)atof(strtok(NULL, DELIMITER));
            }
        }
    #endif
    
    fclose(myfile);
    return retVal;
}

float* ParseSampleInput(char* f){
    int length = strlen(f);
    printf("File Extension: %s\n",f+length-3);
    if(strcmp(f+length-3,"bin") == 0) {
        return readBIN(f);
    } else {
        return readCSV(f);
    }
}


void FreeMatrix(float* d_matrix){
    CUDA_SAFE_CALL(hipFree(d_matrix));
}

float* BuildQGPU(float* d_events, float* d_clusters, float* distanceMatrix, float* mdlTime, int gpu_id, int num_gpus, int my_num_events){
    float* d_matrix;
    int size = sizeof(float) * NUM_CLUSTERS*NUM_CLUSTERS;

    cudaTimer_t timer_gpu;
    cudaTimer_t timer_memcpy;
    createTimer(&timer_gpu);
    createTimer(&timer_memcpy);
    
    startTimer(timer_memcpy);
    hipMalloc((void**)&d_matrix, size);
    printCudaError();
    stopTimer(timer_memcpy);
    
    startTimer(timer_gpu);
    dim3 grid(NUM_CLUSTERS / num_gpus, NUM_CLUSTERS);
    int start_row = gpu_id*(NUM_CLUSTERS/num_gpus);
    printf("GPU %d: Starting row for Q Matrix: %d\n",gpu_id,start_row);

    printf("Launching Q Matrix Kernel\n");
   CalculateQMatrixGPUUpgrade<<<grid, Q_THREADS>>>(d_events, d_clusters, d_matrix, distanceMatrix, start_row, my_num_events);
    hipDeviceSynchronize();
    printCudaError();
    stopTimer(timer_gpu);

    startTimer(timer_memcpy);
    float* matrix = (float*)malloc(size);
    printf("Copying results to CPU\n");
    hipError_t error = hipMemcpy(matrix, d_matrix, size, hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
    printf(hipGetErrorString(hipGetLastError()));
    printf("\n");
    stopTimer(timer_memcpy);

    stopTimer(timer_gpu);
    *mdlTime = getTimerValue(timer_gpu);
    printf("Processing time for MDL GPU: %f (ms) \n", *mdlTime);
    printf("Memcpy time for MDL GPU: %f (ms) \n", getTimerValue(timer_memcpy));
    
    deleteTimer(timer_gpu);
    deleteTimer(timer_memcpy);
        
    printCudaError();
    
    FreeMatrix(d_matrix);
    return matrix;
}
